#include "hip/hip_runtime.h"
// See 'LICENSE_PHANTASY_ENGINE' for copyright and contributors.

#include "kernels/ShadeSecondaryHit.hpp"

#include "hip/hip_math_constants.h"

#include "CudaHelpers.hpp"
#include "CudaPbr.cuh"

namespace phe {

// ShadeSecondaryHitKernel
// ------------------------------------------------------------------------------------------------

static __global__ void shadeSecondaryHitKernel(ShadeSecondaryHitKernelInput input,
                                               IncomingLight* __restrict__ incomingLightsOut)
{
	// Calculate index in array
	uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= input.numRayHitInfos) return;

	RayIn ray = input.secondaryRays[idx];
	RayHitInfo info = input.rayHitInfos[idx];

	vec3 p = info.position();
	vec3 n = info.normal();
	vec3 albedo = info.albedo();
	float roughness = info.roughness();
	float metallic = info.metallic();

	vec3 v = normalize(ray.origin() - p);

	// Interpolation of normals sometimes makes them face away from the camera. Clamp
	// these to almost zero, to not break shading calculations.
	float nDotV = fmaxf(0.001f, dot(n, v));

	vec3 color = vec3(0.0f);

	uint32_t baseShadowIdx = idx * input.numStaticSphereLights;
	for (uint32_t i = 0; i < input.numStaticSphereLights; i++) {
		
		// Check if light source is occluded or not
		bool inLight = input.shadowRayResults[baseShadowIdx + i];
		if (!inLight) continue;

		// Retrieve light source
		SphereLight light = input.staticSphereLights[i];
		vec3 toLight = light.pos - p;
		float toLightDist = length(toLight);
		vec3 l = toLight / toLightDist;

		// Shade
		vec3 shading = shade(p, n, v, albedo, roughness, metallic, l, toLightDist, light.strength, light.range);
		color += shading * fallofFactor(toLightDist, light.range);
	}

	// Calculate incoming light struct to output
	IncomingLight tmp;
	tmp.setOrigin(p);
	tmp.setAmount(color);
	tmp.setFallofFactor(1.0f); // TODO: Assume that we don't need to scale this by distance

	// Calculate write indices and write to the corresponding 4 fullscreen pixels
	vec2i halfRes = input.res / 2;
	vec2i halfResIdx = vec2i(idx % halfRes.x, idx / halfRes.x);
	vec2i fullResIdx = halfResIdx * 2;

	vec2i loc1 = fullResIdx;
	vec2i loc2 = loc1 + vec2i(1, 0);
	vec2i loc3 = loc1 + vec2i(0, 1);
	vec2i loc4 = loc1 + vec2i(1, 1);

	int idx1 = (loc1.y * input.res.x + loc1.x) * input.numIncomingLightsPerPixel;
	int idx2 = (loc2.y * input.res.x + loc2.x) * input.numIncomingLightsPerPixel;
	int idx3 = (loc3.y * input.res.x + loc3.x) * input.numIncomingLightsPerPixel;
	int idx4 = (loc4.y * input.res.x + loc4.x) * input.numIncomingLightsPerPixel;

	incomingLightsOut[idx1] = tmp;
	incomingLightsOut[idx2] = tmp;
	incomingLightsOut[idx3] = tmp;
	incomingLightsOut[idx4] = tmp;
}

// ShadeSecondaryHitKernel launch function
// ------------------------------------------------------------------------------------------------

void launchShadeSecondaryHitKernel(const ShadeSecondaryHitKernelInput& input,
                                   IncomingLight* __restrict__ incomingLightsOut) noexcept
{
	const uint32_t numThreadsPerBlock = 256;
	uint32_t numBlocks = (input.numRayHitInfos / numThreadsPerBlock) + 1;

	shadeSecondaryHitKernel<<<numBlocks, numThreadsPerBlock>>>(input, incomingLightsOut);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

} // namespace phe
