#include "hip/hip_runtime.h"
// See 'LICENSE_PHANTASY_ENGINE' for copyright and contributors.

#include "RayCastKernel.hpp"

#include <phantasy_engine/level/SphereLight.hpp>
#include <phantasy_engine/ray_tracer_common/BVHNode.hpp>
#include <phantasy_engine/ray_tracer_common/Triangle.hpp>
#include <phantasy_engine/ray_tracer_common/Shading.hpp>
#include <phantasy_engine/rendering/Material.hpp>
#include <sfz/math/MathHelpers.hpp>

#include "CudaHelpers.hpp"
#include "CudaSfzVectorCompatibility.cuh"
#include "MaterialKernel.hpp"
#include "GBufferRead.cuh"

namespace phe {

using sfz::vec2;
using sfz::vec2i;
using sfz::vec3;
using sfz::vec3i;
using sfz::vec4;
using sfz::vec4i;


// Material access helpers
// ------------------------------------------------------------------------------------------------

__device__ float readMaterialTextureGray(hipTextureObject_t texture, vec2 coord) noexcept {
	uchar1 res = tex2D<uchar1>(texture, coord.x, coord.y);
	return float(res.x) / 255.0f;
}

__device__ vec4 readMaterialTextureRGBA(hipTextureObject_t texture, vec2 coord) noexcept
{
	uchar4 res = tex2D<uchar4>(texture, coord.x, coord.y);
	return vec4(float(res.x), float(res.y), float(res.z), float(res.w)) / 255.0f;
}

inline __device__ float linearize(float value)
{
	return std::pow(value, 2.2f);
}

struct HitInfo final {
	vec3 pos;
	vec3 normal;
	vec2 uv;
	uint32_t materialIndex;
};

SFZ_CUDA_CALLABLE HitInfo interpretHit(const TriangleData* triDatas, const RayHit& result,
	const RayIn& ray) noexcept
{
	const TriangleData& data = triDatas[result.triangleIndex];
	float u = result.u;
	float v = result.v;

	// Retrieving position
	HitInfo info;
	info.pos = ray.origin() + result.t * ray.dir();

	// Interpolating normal
	vec3 n0 = data.n0;
	vec3 n1 = data.n1;
	vec3 n2 = data.n2;
	info.normal = normalize(n0 + (n1 - n0) * u + (n2 - n0) * v);

	 // Interpolating uv coordinate
	vec2 uv0 = data.uv0;
	vec2 uv1 = data.uv1;
	vec2 uv2 = data.uv2;
	info.uv = uv0 + (uv1 - uv0) * u + (uv2 - uv0) * v;

	// Material index
	info.materialIndex = data.materialIndex;

	return info;
}

__device__ void shadeHit(PathState& pathState, RayIn& shadowRay,
	const vec3& normal, const vec3& toCamera, const vec3& pos, const vec3& offsetPos,
	const vec3& albedoColor, float metallic, float roughness,
	const SphereLight* sphereLights, uint32_t numSphereLights) noexcept
{
	vec3 offsetHitPos = pos + 0.01f * normal;
	// TEMP: Restrict to single light source
	numSphereLights = 2;
	for (uint32_t i = 1; i < numSphereLights; i++) {
		SphereLight light = sphereLights[i];

		vec3 toLight = light.pos - pos;
		float toLightDist = length(toLight);
		vec3 l = toLight / toLightDist;
		vec3 v = normalize(toCamera);
		vec3 h = normalize(l + v);

		float nDotL = dot(normal, l);
		if (nDotL <= 0.0f) {
			continue;
		}

		float nDotV = dot(normal, v);

		nDotV = std::max(0.001f, nDotV);

		// Lambert diffuse
		vec3 diffuse = albedoColor / sfz::PI();

		// Cook-Torrance specular
		// Normal distribution function
		float nDotH = std::max(sfz::dot(normal, h), 0.0f); // max() should be superfluous here
		float ctD = ggx(nDotH, roughness * roughness);

		// Geometric self-shadowing term
		float k = pow(roughness + 1.0f, 2.0f) / 8.0f;
		float ctG = geometricSchlick(nDotL, nDotV, k);

		// Fresnel function
		// Assume all dielectrics have a f0 of 0.04, for metals we assume f0 == albedo
		vec3 f0 = sfz::lerp(vec3(0.04f), albedoColor, metallic);
		vec3 ctF = fresnelSchlick(nDotL, f0);

		// Calculate final Cook-Torrance specular value
		vec3 specular = ctD * ctF * ctG / (4.0f * nDotL * nDotV);

		// Calculates light strength
		float fallofNumerator = pow(sfz::clamp(1.0f - std::pow(toLightDist / light.range, 4.0f), 0.0f, 1.0f), 2);
		float fallofDenominator = (toLightDist * toLightDist + 1.0f);
		float falloff = fallofNumerator / fallofDenominator;
		vec3 lighting = falloff * light.strength;

		vec3 color = (diffuse + specular) * lighting * nDotL;

		if (light.staticShadows) {
			// Slightly offset light ray to get stochastic soft shadows
			vec3 circleU;
			if (abs(normal.z) > 0.01f) {
				circleU = normalize(vec3(0.0f, -normal.z, normal.y));
			}
			else {
				circleU = normalize(vec3(-normal.y, normal.x, 0.0f));
			}
			vec3 circleV = cross(circleU, toLight);

			// TODO: Use RNG
			float r1 = 0.5f;
			float r2 = (2.0f * light.radius * 0.5f) - light.radius;
			float azimuthAngle = 2.0f * sfz::PI() * r1;

			vec3 lightPosOffset = circleU * cos(azimuthAngle) * r2 +
				circleV * sin(azimuthAngle) * r2;

			vec3 offsetLightDiff = light.pos + lightPosOffset - offsetHitPos;
			vec3 offsetLightDir = normalize(offsetLightDiff);

			shadowRay.setOrigin(offsetHitPos);
			shadowRay.setDir(offsetLightDir);
			shadowRay.setMaxDist(length(offsetLightDiff));
			shadowRay.setNoResultOnlyHit(true);

			pathState.pendingLightContribution = color;
		}
		else {
			pathState.finalColor += color;
		}
	}
}

static __global__ void materialKernel(
	vec2i res,
	RayIn* shadowRays,
	PathState* pathStates,
	const RayIn* rays,
	const RayHit* rayHits,
	const TriangleData* staticTriangleDatas,
	const Material* materials,
	const hipTextureObject_t* textures,
	const SphereLight* sphereLights,
	uint32_t numSphereLights)
{
	// Calculate surface coordinates
	vec2i loc = vec2i(blockIdx.x * blockDim.x + threadIdx.x,
	                  blockIdx.y * blockDim.y + threadIdx.y);
	if (loc.x >= res.x || loc.y >= res.y) return;

	uint32_t id = loc.y * res.x + loc.x;
	PathState& pathState = pathStates[id];
	RayIn& shadowRay = shadowRays[id];
	RayHit hit = rayHits[id];
	RayIn ray = rays[id];

	if (hit.triangleIndex == ~0u) {
		return;
	}

	HitInfo info = interpretHit(staticTriangleDatas, hit, ray);

	const Material& material = materials[info.materialIndex];
	vec4 albedoValue = material.albedoValue();
	if (materials[info.materialIndex].albedoTexIndex() != UINT32_MAX) {
		hipTextureObject_t albedoTexture = textures[material.albedoTexIndex()];
		albedoValue = readMaterialTextureRGBA(albedoTexture, info.uv);
	}
	vec3 albedoColor = albedoValue.xyz;
	albedoColor = linearize(albedoColor);

	float metallic = material.metallicValue();
	if (materials[info.materialIndex].metallicTexIndex() != UINT32_MAX) {
		hipTextureObject_t metallicTexture = textures[material.metallicTexIndex()];
		metallic = readMaterialTextureGray(metallicTexture, info.uv);
	}
	metallic = linearize(metallic);

	float roughness = material.roughnessValue();
	if (materials[info.materialIndex].roughnessTexIndex() != UINT32_MAX) {
		hipTextureObject_t roughnessTexture = textures[material.roughnessTexIndex()];
		roughness = readMaterialTextureGray(roughnessTexture, info.uv);
	}
	roughness = linearize(roughness);

	vec3 offsetHitPos = info.pos + info.normal * 0.01f;

	shadeHit(pathState, shadowRay, info.normal, -ray.dir(), info.pos, offsetHitPos, albedoColor, metallic, roughness, sphereLights, numSphereLights);
}

void launchMaterialKernel(const MaterialKernelInput& input) noexcept
{
	// Calculate number of threads and blocks to run
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((input.res.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(input.res.y + threadsPerBlock.y - 1) / threadsPerBlock.y);

	// Run cuda ray tracer kernel
	materialKernel<<<numBlocks, threadsPerBlock>>>(input.res, input.shadowRays, input.pathStates, input.rays, input.rayHits, input.staticTriangleDatas, input.materials, input.textures, input.sphereLights, input.numSphereLights);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

static __global__ void materialPrimaryKernel(
	vec2i res,
	vec3 camPos,
	RayIn* shadowRays,
	PathState* pathStates,
	hipSurfaceObject_t posTex,
	hipSurfaceObject_t normalTex,
	hipSurfaceObject_t albedoTex,
	hipSurfaceObject_t materialTex,
	const SphereLight* sphereLights,
	uint32_t numSphereLights)
{
	// Calculate surface coordinates
	vec2i loc = vec2i(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	if (loc.x >= res.x || loc.y >= res.y) return;

	GBufferValue gBufferValue = readGBuffer(posTex, normalTex, albedoTex, materialTex, loc);

	vec3 offsetHitPos = gBufferValue.pos + gBufferValue.normal * 0.01f;

	uint32_t id = loc.y * res.x + loc.x;
	PathState& pathState = pathStates[id];
	RayIn& shadowRay = shadowRays[id];

	vec3 toCamera = camPos - gBufferValue.pos;

	shadeHit(pathState, shadowRay, gBufferValue.normal, toCamera, gBufferValue.pos, offsetHitPos, gBufferValue.albedo, gBufferValue.metallic, gBufferValue.roughness, sphereLights, numSphereLights);
}

void launchGBufferMaterialKernel(const GBufferMaterialKernelInput& input) noexcept
{
	// Calculate number of threads and blocks to run
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((input.res.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
	               (input.res.y + threadsPerBlock.y - 1) / threadsPerBlock.y);

	// Run cuda ray tracer kernel
	materialPrimaryKernel<<<numBlocks, threadsPerBlock>>>(input.res, input.camPos, input.shadowRays, input.pathStates, input.posTex, input.normalTex, input.albedoTex, input.materialTex, input.sphereLights, input.numSphereLights);

	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

static __global__ void initPathStates(vec2i res, PathState* pathStates)
{
	// Calculate surface coordinates
	vec2i loc = vec2i(blockIdx.x * blockDim.x + threadIdx.x,
	                  blockIdx.y * blockDim.y + threadIdx.y);
	if (loc.x >= res.x || loc.y >= res.y) return;

	// Read rayhit from array
	uint32_t id = loc.y * res.x + loc.x;

	PathState& pathState = pathStates[id];
	pathState.finalColor = vec3(0.0f);
	pathState.pathLength = 0;
	pathState.pendingLightContribution = vec3(0.0f);
}

void launchInitPathStatesKernel(vec2i res, PathState* pathStates) noexcept
{
	// Calculate number of threads and blocks to run
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((res.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
	               (res.y + threadsPerBlock.y - 1) / threadsPerBlock.y);

	// Run cuda ray tracer kernel
	initPathStates<<<numBlocks, threadsPerBlock>>>(res, pathStates);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

static __global__ void writeResultKernel(hipSurfaceObject_t surface, vec2i res,
                                         const RayHit* shadowRayHits, PathState* pathStates)
{
	// Calculate surface coordinates
	vec2i loc = vec2i(blockIdx.x * blockDim.x + threadIdx.x,
	                  blockIdx.y * blockDim.y + threadIdx.y);
	if (loc.x >= res.x || loc.y >= res.y) return;

	uint32_t id = loc.y * res.x + loc.x;

	PathState& pathState = pathStates[id];
	const RayHit& shadowRayHit = shadowRayHits[id];
	if (shadowRayHit.triangleIndex == UINT32_MAX) {
		pathState.finalColor = pathState.pendingLightContribution;
	}

	vec4 color4 = vec4(pathState.finalColor, 1.0f);
	surf2Dwrite(toFloat4(color4), surface, loc.x * sizeof(float4), loc.y);
}

void launchWriteResultKernel(const WriteResultKernelInput& input) noexcept
{
	// Calculate number of threads and blocks to run
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks((input.res.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
	               (input.res.y + threadsPerBlock.y - 1) / threadsPerBlock.y);

	// Run cuda ray tracer kernel
	writeResultKernel<<<numBlocks, threadsPerBlock>>>(input.surface, input.res, input.rayHits, input.pathStates);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

} // namespace phe
