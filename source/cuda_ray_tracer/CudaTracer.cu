#include "hip/hip_runtime.h"
// See 'LICENSE_PHANTASY_ENGINE' for copyright and contributors.

#include "CudaTracer.cuh"

#include <math.h>

#include <sfz/math/Vector.hpp>

#include "CudaHelpers.hpp"
#include "CudaSfzVectorCompatibility.cuh"

namespace phe {

using namespace sfz;

inline __device__ void writeSurface(const hipSurfaceObject_t& surface, vec2i loc, const vec4& data) noexcept
{
	float4 dataFloat4 = toFloat4(data);
	surf2Dwrite(dataFloat4, surface, loc.x * sizeof(float4), loc.y);
}

inline __device__ vec3 calculateRayDir(const CameraDef& cam, vec2 loc, vec2 surfaceRes) noexcept
{
	vec2 locNormalized = loc / surfaceRes; // [0, 1]
	vec2 centerOffsCoord = locNormalized * 2.0f - vec2(1.0f); // [-1.0, 1.0]
	centerOffsCoord.y = -centerOffsCoord.y;
	vec3 nonNormRayDir = cam.dir + cam.dX * centerOffsCoord.x + cam.dY * centerOffsCoord.y;
	return normalize(nonNormRayDir);
}

__global__ void cudaRayTracerKernel(hipSurfaceObject_t surface, vec2i surfaceRes, CameraDef cam,
                                    StaticSceneCuda staticScene)
{
	// Calculate surface coordinates
	vec2i loc = vec2i(blockIdx.x * blockDim.x + threadIdx.x,
	                  blockIdx.y * blockDim.y + threadIdx.y);
	if (loc.x >= surfaceRes.x || loc.y >= surfaceRes.y) return;

	// Calculate ray direction
	vec3 rayDir = calculateRayDir(cam, vec2(loc), vec2(surfaceRes));
	Ray ray(cam.origin, rayDir);

	// Ray cast against BVH
	RayCastResult hit = castRay(staticScene.bvhNodes, staticScene.triangleVertices, ray);
	if (hit.triangleIndex == ~0u) {
		writeSurface(surface, loc, vec4(0.0f));
		return;
	}

	HitInfo info = interpretHit(staticScene.triangleDatas, hit, ray);
	vec3 lightPos = staticScene.pointLights[0].pos;
	vec3 l = -normalize(info.pos - lightPos);

	writeSurface(surface, loc, vec4(vec3(dot(l, info.normal)), 1.0));
}

void runCudaRayTracer(hipSurfaceObject_t surface, vec2i surfaceRes, const CameraDef& cam,
                      const StaticSceneCuda& staticScene) noexcept
{	
	// Calculate number of threads and blocks to run
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((surfaceRes.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
	               (surfaceRes.y + threadsPerBlock.y  - 1) / threadsPerBlock.y);

	// Run cuda ray tracer kernel
	cudaRayTracerKernel<<<numBlocks, threadsPerBlock>>>(surface, surfaceRes, cam, staticScene);
	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

} // namespace phe
