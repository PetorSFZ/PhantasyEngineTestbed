#include "hip/hip_runtime.h"
// See 'LICENSE_PHANTASY_ENGINE' for copyright and contributors.

#include "CUDATest.cuh"

__global__ void writeBlauImpl(hipSurfaceObject_t surf, int width, int height)
{
	// Calculate surface coordinates
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		float4 data = make_float4(x / float(width), y / float(height), 1.0f, 1.0f);
		surf2Dwrite(data, surf, x * 4 * 4, y);
	}
}

namespace phe {

void writeBlau(hipSurfaceObject_t surf, vec2i surfRes, vec2i currRes) noexcept
{
	int width = surfRes.x;
	int height = surfRes.y;
	
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
	               (height + threadsPerBlock.y  - 1) / threadsPerBlock.y);

	writeBlauImpl<<<numBlocks, threadsPerBlock>>>(surf, width, height);
}

}
