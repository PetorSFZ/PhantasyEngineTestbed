#include "hip/hip_runtime.h"
// See 'LICENSE_PHANTASY_ENGINE' for copyright and contributors.

#include "CudaTracerEntry.cuh"

#include <sfz/math/Vector.hpp>

#include "CudaVectorHelpers.cuh"

#include <math.h>

struct CameraDefCuda final {
	float3 origin;
	float3 dir; // normalized
	float3 up; // normalized and orthogonal to camDir
	float3 right; // normalized and orthogonal to both camDir and camUp
	float vertFovRad;
};

inline __device__ float3 calculateRayDir(const CameraDefCuda& cam, float2 loc, float2 surfaceRes) noexcept
{
	float2 locNormalized = loc / surfaceRes; // [0, 1]
	float2 centerOffsCoord = locNormalized * 2.0f - 1.0f; // [-1.0, 1.0]

	// Move out
	float alphaMaxX = atan(cam.vertFovRad) / 2.0f;
	float aspect = surfaceRes.x / surfaceRes.y;
	float alphaMaxY = aspect * alphaMaxX;
	float3 UP = cam.up * alphaMaxY;
	float3 RIGHT = cam.right * alphaMaxX;

	float3 nonNormRayDir = cam.dir + RIGHT * centerOffsCoord.x + UP * centerOffsCoord.y;
	return normalize(nonNormRayDir);
}

__global__ void cudaRayTracerKernel(hipSurfaceObject_t surface, int2 surfaceRes, CameraDefCuda cam)
{
	// Calculate surface coordinates
	int2 loc = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
	                     blockIdx.y * blockDim.y + threadIdx.y);
	if (loc.x >= surfaceRes.x || loc.y >= surfaceRes.y) return;

	// Calculate ray direction
	float3 rayDir = calculateRayDir(cam, toFloat2(loc), toFloat2(surfaceRes));

	// Write ray dir to texture for now
	float4 data = make_float4(rayDir.x, rayDir.y, rayDir.z, 1.0f);
	surf2Dwrite(data, surface, loc.x * sizeof(float4), loc.y);

}

namespace phe {

using sfz::vec3;

void runCudaRayTracer(hipSurfaceObject_t surface, vec2i surfaceRes, const CameraDef& cam) noexcept
{
	// Convert camera defintion to CUDA primitives
	CameraDefCuda camTmp;
	camTmp.origin = toFloat3(cam.origin);
	camTmp.dir = toFloat3(cam.dir);
	camTmp.up = toFloat3(cam.up);
	camTmp.right = toFloat3(cam.right);
	camTmp.vertFovRad = cam.vertFovRad;

	// Calculate number of threads and blocks to run
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((surfaceRes.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
	               (surfaceRes.y + threadsPerBlock.y  - 1) / threadsPerBlock.y);

	// Run cuda ray tracer kernel
	cudaRayTracerKernel<<<numBlocks, threadsPerBlock>>>(surface, toInt2(surfaceRes), camTmp);
	hipDeviceSynchronize();
}

} // namespace phe
